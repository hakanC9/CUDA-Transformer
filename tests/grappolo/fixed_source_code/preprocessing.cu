// ***********************************************************************
//
//     Rundemanen: CUDA C++ parallel program for community detection
//   Md Naim (naim.md@gmail.com), Fredrik Manne (Fredrik.Manne@uib.no)
//                       University of Bergen
//
// ***********************************************************************
//
//       Copyright (2016) University of Bergen
//                      All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//
// 1. Redistributions of source code must retain the above copyright
// notice, this list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright
// notice, this list of conditions and the following disclaimer in the
// documentation and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its
// contributors may be used to endorse or promote products derived from
// this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
// "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
// LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
// FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
// COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
// INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
// BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
// LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
// ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
// POSSIBILITY OF SUCH DAMAGE.
//
// ************************************************************************

#include <algorithm>
#include <iostream>
#include "communityGPU.h"
#include"hostconstants.h"
#include <thrust/count.h>
#include "thrust/host_vector.h"

void Community::preProcess() {



    //Compute degree of each node
    thrust::device_vector<int> sizesOfNhoods(g.indices.size() - 1, 0);


    thrust::transform(g.indices.begin() + 1, g.indices.end(), g.indices.begin(),
            sizesOfNhoods.begin(), thrust::minus<int >());

    //Find all degree 1 vertices
    IsInRange<int, int> filter_SNL_1(1, 1);

    int nrC_SNL_1 = thrust::count_if(thrust::device, sizesOfNhoods.begin(),
            sizesOfNhoods.end(), filter_SNL_1);

    std::cout << "#vertices (SNL=1):" << nrC_SNL_1 << std::endl;

    //Lets copy Identities of all communities  in g_next.links

    g_next.links.resize(community_size, 0);
    thrust::sequence(g_next.links.begin(), g_next.links.end(), 0);

    //Use g_next.indices to copy community ids with  SLN =1
    g_next.indices.resize(community_size, -1);


    //Collet all  degree 1 vertices in g_next.indices
    thrust::copy_if(thrust::device, g_next.links.begin(), g_next.links.end(),
            sizesOfNhoods.begin(), g_next.indices.begin(), filter_SNL_1);



    /*
    void reduceGraph(int* indices, unsigned int* links, float* weights, int gType,
            int* uniDegvrts, unsigned int nrUniDegVrts, unsigned int mark,
            int* vtsForPostProcessing);
     */


    int mark = g.nb_nodes * 2;
    thrust::device_vector<int> vtsForPostProcessing(nrC_SNL_1, -1);

    unsigned int nrBlk = (nrC_SNL_1 + NR_THREAD_PER_BLOCK - 1) / NR_THREAD_PER_BLOCK;

    //initialization of n2c
    n2c.resize(community_size);
    thrust::sequence(n2c.begin(), n2c.end(), 0);
    if(nrC_SNL_1>0)
    reduceGraph <<<nrBlk, NR_THREAD_PER_BLOCK>>>(
            thrust::raw_pointer_cast(g.indices.data()),
            thrust::raw_pointer_cast(g.links.data()),
            thrust::raw_pointer_cast(g.weights.data()), g.type,
            thrust::raw_pointer_cast(g_next.indices.data()), nrC_SNL_1, mark,
            thrust::raw_pointer_cast(vtsForPostProcessing.data()),
            thrust::raw_pointer_cast(n2c.data()));

    /*
    // let's process each vertex in  vtsForPostProcessing with a warp
    nrBlk = (nrC_SNL_1 + (NR_THREAD_PER_BLOCK / PHY_WRP_SZ) - 1) / (NR_THREAD_PER_BLOCK / PHY_WRP_SZ);

    void editEdgeList(int* indices, unsigned int* links, float* weights, int gType,
            int* uniDegvrts, unsigned int nrUniDegVrts, unsigned int mark,
            int* vtsForPostProcessing);

    editEdgeList <<<nrBlk, NR_THREAD_PER_BLOCK>>>(
            thrust::raw_pointer_cast(g.indices.data()),
            thrust::raw_pointer_cast(g.links.data()),
            thrust::raw_pointer_cast(g.weights.data()), g.type,
            thrust::raw_pointer_cast(g_next.indices.data()), nrC_SNL_1, mark,
            thrust::raw_pointer_cast(vtsForPostProcessing.data()));
     */


    if (0) {
        thrust::host_vector<int> uniDegVertices = g_next.indices;
        std::cout << std::endl;
        for (int i = 0; i < nrC_SNL_1; i++) {
            std::cout << uniDegVertices[i] << " ";
        }
        std::cout << std::endl;
    }

    if (0) {
        thrust::host_vector<unsigned int> gnlinks = g.links;
        thrust::host_vector<int> gnIndices = g.indices;
        thrust::device_vector<float> gnWeights = g.weights;
        for (unsigned int i = 0; i < g.nb_nodes; i++) {

            unsigned int startNbr = gnIndices[i];
            unsigned int endNbr = gnIndices[i + 1];
            //thrust::sort(gnlinks.begin() + startNbr, gnlinks.begin() + endNbr);

           if(i<10){
	    std::cout << i << ":";

            for (unsigned int j = startNbr; j < endNbr; j++) {

                float edgeWt = 1;
                if (g.type == WEIGHTED)
                    edgeWt = gnWeights[j];
                std::cout << " " << gnlinks[j] << "(" << edgeWt << ")";
            }
            std::cout << std::endl;
	    }
        }
    }

    n2c.clear();
    vtsForPostProcessing.clear();
    sizesOfNhoods.clear();
    g_next.links.clear();
    g_next.indices.clear();

}
